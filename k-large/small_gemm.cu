//
// 64 * 64 floats
// 2 float2 per workitem
// 16 * 16 workitems per workgroup, each working on 4 floats
// this makes up to 32 * 32 floats
// 4 workgroups gets launched
//

#include <iostream>
#include <vector>
#include <chrono>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

__global__ void Kernel(float2* matrix_a, float2* matrix_b, float2* matrix_c) {
    unsigned tx = threadIdx.x;
    unsigned ty = threadIdx.y;

    unsigned bx = blockIdx.x;
    unsigned by = blockIdx.y;

    unsigned index_a = tx + bx * 16;
    unsigned index_b = ty + by * 16;

    unsigned index_c0 = tx + bx * 16 + ty * 32 * 2 + by * 2 * 32 * 16;
    unsigned index_c1 = index_c0 + 32;
    float2 c0 = matrix_c[index_c0];
    float2 c1 = matrix_c[index_c1];

    for(size_t k = 0; k < 64; k++) {
    float2 a = matrix_a[index_a + k * 32];
    float2 b = matrix_b[index_b + k * 32];

    c0.x += a.x * b.x;
    c0.y += a.x * b.y;
    c1.x += a.y * b.x;
    c1.y += a.y * b.y;

    }

    matrix_c[index_c0] = c0;
    matrix_c[index_c1] = c1;
}

int main() {
    size_t m = 64, n = 64, k = 64;
    size_t num_iter = 1;
    size_t size = m * n * sizeof(float);
    std::vector<float> A(m * k);
    std::vector<float> B(n * k);
    std::vector<float> C(n * m);

    std::fill(A.begin(), A.end(), 1.0f);
    std::fill(B.begin(), B.end(), 2.0f);
    std::fill(C.begin(), C.end(), 1.0f);;

    float2* Ad, *Bd, *Cd;
    hipMalloc(&Ad, size);
    hipMalloc(&Bd, size);
    hipMalloc(&Cd, size);

    hipMemcpy(Ad, A.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(Bd, B.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(Cd, C.data(), size, hipMemcpyHostToDevice);

    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

    for(size_t i = 0; i < num_iter; i++) {
    Kernel<<<dim3(2,2,1), dim3(16,16,1)>>>(Ad, Bd, Cd);
    }
    hipDeviceSynchronize();

    std::chrono::high_resolution_clock::time_point stop = std::chrono::high_resolution_clock::now();

    double time = std::chrono::duration_cast<std::chrono::duration<double>>(stop - start).count();

    std::cout << time << std::endl;

    hipMemcpy(C.data(), Cd, size, hipMemcpyDeviceToHost);

    for(size_t i = 0; i < n; i++) {
        for(size_t j = 0; j < m; j++) {
            std::cout << C[i + j * m] << " ";
        }
        std::cout << std::endl;
    }

}

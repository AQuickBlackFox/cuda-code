
#include <hip/hip_runtime.h>
#include<iostream>
#include <stdio.h>

#define ITER 1024*1024*16
#define WI 512

#include <time.h>
#include <sys/time.h>
#define USECPSEC 1000000ULL

unsigned long long dtime_usec(unsigned long long start){

  timeval tv;
  gettimeofday(&tv, 0);
  return ((tv.tv_sec*USECPSEC)+tv.tv_usec)-start;
}

__global__ void XorPopCntAccum(int *in1d, int* outd) {
    int tx = threadIdx.x;
    int in1 = in1d[tx];
    int out = outd[tx];
    for (int i = 0; i < ITER; i++) {
      out += __popc((out ^ in1));
    }
    outd[tx] = out;
}


int main() {
    hipDeviceProp_t prop;
    unsigned device = 0;
    hipSetDevice(device);
    hipGetDeviceProperties(&prop, device);
    std::cout<<prop.name<<std::endl;
    unsigned CU = prop.multiProcessorCount;
    int *in1d, *outd;
    hipMalloc((void**)&in1d, WI*4);
    hipMalloc((void**)&outd, WI*4);
    XorPopCntAccum<<<1, WI>>>(in1d, outd);  // warm up
    hipDeviceSynchronize();
    unsigned long long dt = dtime_usec(0);
    XorPopCntAccum<<<CU, WI>>>(in1d, outd);
    hipDeviceSynchronize();
    dt = dtime_usec(dt);
    unsigned long long ops = ITER;
    ops *= CU;
    ops *= WI;
    float et = dt/(float)USECPSEC;
    unsigned long long Mops = ops/1000000;
    std::cout<<et<<"s for "<< Mops << " XOR+POPCNT+ACCUM"<<std::endl;
    float tp = (Mops)/(et*1000000);
    std::cout << "throughput: " << tp << " Tops/s" << std::endl;
}
